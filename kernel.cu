#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>

#define BLOCK_SIZE 16 // submatrix size
#define N 1024 // matrix size is N*N

typedef struct {
    int n;
    int* elements;
} Matrix;

__device__ float GetElement(const Matrix A, int row, int col) {
    return A.elements[row * N + col];
}

__device__ void SetElement(Matrix A, int row, int col, int value) {
    A.elements[row * N + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col, int n) {
    Matrix ASub;
    ASub.n = BLOCK_SIZE;
    ASub.elements = &A.elements[n * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return ASub;
}

__global__ void matMulKernel(int* a, int* b, int* c, int n) {
    // ���������� ����� � ������� (����� ����� � ������ � � �������)
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    // ������ ���� ����� ��������� ���� ���������� Csub, ��� ����
    // ������ ���� ������ ���� ����������� ���������� ������� Csub
    Matrix cc;
    cc.elements = c;
    Matrix aa;
    aa.elements = a;
    Matrix bb;
    bb.elements = b;

    Matrix Csub = GetSubMatrix(cc, blockRow, blockCol,n);
    // ������ ���� ��������� ���� ������� ���������� Csub
    float Cvalue = 0;
    // thread row and col WITHIN CSUB
    int row = threadIdx.y;
    int col = threadIdx.x;
    // ���� �� ���� ����������� ������ ������ A � ������� ������ B;
    // ���� ���� ��������� ��� ���������� ���� ���������� Csub.
    // ������� ��������� ���� ��������� � ����������� ����������
    for (int m = 0; m < (N / BLOCK_SIZE); ++m) {
        // ������������ ������������ ��������� Asub � Bsub
        Matrix Asub = GetSubMatrix(aa, blockRow, m,n);
        Matrix Bsub = GetSubMatrix(bb, m, blockCol,n);
        // ����������� ��������� ASub � Bsub � ����������� ������
        // ������ ���� ��������� ���� ������� ASub � ���� � Bsub
        // �������� ��������: ������ ���� ��������� ������� As � Bs,
        // ���� ���� ����� �������� ������ ���� ������� As � ���� Bs
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);
        // ���� ���������������� � ����� ���������, ��� �� ���������
        __syncthreads();
        // ��������� ������������ ����� ������ Asub � ������ ������� Bsub
        // ��������� (��������) ���� ������� �������������� ����������
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
        // ���� ���������, ��� ��� Cvalues �������������� �� ������
        // ���������� ����������� ������ � ����������� ������� As, Bs
        __syncthreads();
    }
    // ������ Csub � ���������� ������: ������ ���� ���������� ����� �������
    SetElement(Csub, row, col, Cvalue);
}

//__global__ void matMulKernel(float* a, float* b, float* c, int n) {
    // ������ ���� ��������� ���� ������� ������� C
  //  float cvalue = 0;
 //   int row = blockIdx.y * blockDim.y + threadIdx.y;
 //   int col = blockIdx.x * blockDim.x + threadIdx.x;
  //  for (int e = 0; e < n; ++e)
  //      cvalue += a[row * n + e] * b[e * n + col];
  //  c[row * n + col] = cvalue;
//}

int** matMulCPU(int** a, int** b, int** c, int n) {
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
        {
            c[i][j] = 0;
            for (int k = 0; k < n; k++)
                c[i][j] += a[i][k] * b[k][j];
        }
    return c;
}

bool checkResult(int** a, int* b, int n) {
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            if (a[i][j] != b[N * i + j])
                return false;
    return true;
}

int main(int argc, char* argv[])
{
    srand(time(NULL));

    int numBytes = N * N * sizeof(int);

    // ��������� ������ �� �����
    int** a_c;
    int** b_c;
    int** c_c;
    a_c = new int* [N];
    for (int i = 0; i < N; i++)
        a_c[i] = new int[N];
    
    b_c = new int* [N];
    for (int i = 0; i < N; i++)
        b_c[i] = new int[N];
    
    c_c = new int* [N];
    for (int i = 0; i < N; i++)
        c_c[i] = new int[N];

    int* a = new int[N * N];
    int* b = new int[N * N];
    int* c = new int[N * N];

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
        {
            int k = N * i + j;
           // a[k] = (rand())%100;
           // b[k] = (rand()) % 100;
            a[k] = (rand()) % 100;
            b[k] = (rand()) % 100;
            a_c[i][j] = a[k];
            b_c[i][j] = b[k];
            c_c[i][j] = 0;
        }

    clock_t time;
    time = clock();
    c_c=matMulCPU(a_c, b_c, c_c, N);
    time = clock() - time;
    //std::cout << static_cast<float>(time) / CLOCKS_PER_SEC << std::endl;
    //std::cout << c_c[0][2] << std::endl;
    //std::cout << c_c[2][7];
    // ��������� ������ �� �������

    int* adev = NULL;
    int* bdev = NULL;
    int* cdev = NULL;

    hipMalloc((void**)&adev, numBytes);
    hipMalloc((void**)&bdev, numBytes);
    hipMalloc((void**)&cdev, numBytes);

    // ��������� ������������ ������� ����

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / threads.x, N / threads.y);

    // �������� ����������� ������� CUDA

    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ���������� ������ ������ �� GPU (��� � ����� 0)

    hipEventRecord(start, 0);
    hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

    matMulKernel << <blocks, threads >> > (adev, bdev, cdev, N);
    //matMulKernel << <blocks, threads >> > (adev, bdev, cdev,N);

    hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    // �������� ����� ������ �� GPU � CPU
    //std::cout << c[2] <<" "<<c[3] <<std::endl;
    if (checkResult(c_c, c, N)) {
        printf("Time spent executing by the GPU: %.2f millseconds\n", gpuTime);
        std::cout << "Time spent executing by the CPU: " << static_cast<float>(time) / CLOCKS_PER_SEC << std::endl;

    }
        
    // ������������ ��������

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);



    delete a;
    delete b;
    delete c;
    for (int i = 0; i < N; i++)
    {
        delete b_c[i];
        delete a_c[i];
        delete c_c[i];
    }
    delete[]a_c;
    delete[]b_c;
    delete[]c_c;

    return 0;

}